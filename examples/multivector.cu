#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------
 *
 *   Copyright (C) 2016 Antonio Augusto Alves Junior
 *
 *   This file is part of Hydra Data Analysis Framework.
 *
 *   Hydra is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   Hydra is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with Hydra.  If not, see <http://www.gnu.org/licenses/>.
 *
 *---------------------------------------------------------------------------*/

/*
 * multivector.cpp
 *
 *  Created on: 19/10/2016
 *      Author: Antonio Augusto Alves Junior
 */

#include <chrono>
#include <time.h>

#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/random.h>

#include <hydra/detail/Config.h>
#include <thrust/device_malloc_allocator.h>
#include <thrust/host_vector.h>
#include <hydra/multivector.h>



using namespace std;
using namespace hydra;





struct AccessOneA
{
	template<typename T>
	__host__ __device__ void operator()(T& e)
	{
		thrust::default_random_engine rng(thrust::default_random_engine::default_seed);
		thrust::uniform_real_distribution<float> UniRng(0.0f, 1.0f);

		e = UniRng(rng);

	}

};

template<unsigned int I >
struct AccessOneB
{
	template<typename T>
	__host__ __device__ void operator()(T& e)
	{
		thrust::default_random_engine rng(thrust::default_random_engine::default_seed);
		thrust::uniform_real_distribution<float> UniRng(0.0f, 1.0f);

		float x = UniRng(rng);
		thrust::get<I>(e) = x;

	}

};


struct AccessAllA
{

	template<typename T>
	__host__ __device__ void operator()(T t)
	{
		thrust::default_random_engine rng(thrust::default_random_engine::default_seed);
		thrust::uniform_real_distribution<float> UniRng(0.0f, 1.0f);


		float x= thrust::get<3>(t) + 2*UniRng(rng);
		float y= thrust::get<4>(t) + 2*UniRng(rng);
		float z= thrust::get<5>(t) + 2*UniRng(rng);

		float r     = sqrt( x*x + y*y + z*z);
		float theta = acos(z/r);
		float phi   = atan(y/x);

		thrust::get<0>(t) = r;
		thrust::get<1>(t) = theta;
		thrust::get<2>(t) = phi;

		thrust::get<3>(t) = x;
		thrust::get<4>(t) = y;
		thrust::get<5>(t) = z;

	}

};

struct AccessAllB
{

	template<typename T>
	__host__ __device__ void operator()(T& t)
	{
		thrust::default_random_engine rng(thrust::default_random_engine::default_seed);
		thrust::uniform_real_distribution<float> UniRng(0.0f, 1.0f);


		float x= thrust::get<3>(t) + 2*UniRng(rng);
		float y= thrust::get<4>(t) + 2*UniRng(rng);
		float z= thrust::get<5>(t) + 2*UniRng(rng);

		float r     = sqrt( x*x + y*y + z*z);
		float theta = acos(z/r);
		float phi   = atan(y/x);

		thrust::get<0>(t) = r;
		thrust::get<1>(t) = theta;
		thrust::get<2>(t) = phi;

		thrust::get<3>(t) = x;
		thrust::get<4>(t) = y;
		thrust::get<5>(t) = z;

	}

};


template<int I=0, typename T>
inline float _for_each_AccessOne1(T& storage)
{
	auto begin = storage.template vbegin<I>();
	auto end   = storage.template vend<I>();

	auto start1 = std::chrono::high_resolution_clock::now();
	thrust::for_each( begin, end, AccessOneA() );
	auto end1 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float, std::milli> elapsed1 = end1 - start1;
	return elapsed1.count();
}

template<int I=0,typename T>
inline float _for_each_AccessOne2(T& storage)
{
	auto start1 = std::chrono::high_resolution_clock::now();
	thrust::for_each(storage.begin(), storage.end(), AccessOneB<I>() );
	auto end1 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float, std::milli> elapsed1 = end1 - start1;
	return elapsed1.count();
}


template<typename T>
inline float _for_each1(T& storage)
{
	auto start1 = std::chrono::high_resolution_clock::now();
	thrust::for_each(storage.begin(), storage.end(), AccessAllA() );
	auto end1 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float, std::milli> elapsed1 = end1 - start1;
	return elapsed1.count();
}

template<typename T>
inline float _for_each2(T& storage)
{
	auto start1 = std::chrono::high_resolution_clock::now();
	thrust::for_each(storage.begin(), storage.end(), AccessAllB() );
	auto end1 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float, std::milli> elapsed1 = end1 - start1;
	return elapsed1.count();
}



size_t n=10000000;

int main(int argv, char** argc)
{

	typedef multivector<thrust::device_vector,
			thrust::device_malloc_allocator,float, float, float,float, float, float> table_t;

	typedef thrust::device_vector<thrust::tuple<float, float, float, float, float, float>> vector_t;
    thrust::tuple<float, float, float, float, float, float> init(0.0,0.0,0.0,2.0,2.0,2.0);

	{


		table_t  storage(n, init);
		float t=_for_each1(storage);
		std::cout << "--------------------------------------------------------------"<<std::endl;
		std::cout << "| multivector (acces all) "<<std::endl;
		std::cout << "| Time (ms) = "<< t <<std::endl;//elapsed1.count() <<std::endl;
		std::cout << "--------------------------------------------------------------"<<std::endl;
		 t=_for_each_AccessOne1<1, table_t >(storage);
		std::cout << "--------------------------------------------------------------"<<std::endl;
		std::cout << "| multivector (access one)"<<std::endl;
		std::cout << "| Time (ms) = "<< t <<std::endl;//elapsed1.count() <<std::endl;
		std::cout << "--------------------------------------------------------------"<<std::endl;

		for(size_t i=0; i<10; i++)
			std::cout<< storage[i] << std::endl;



	}

	//---
	{
		vector_t  storage(n, init);
		//start time
		//auto start1 = std::chrono::high_resolution_clock::now();
		float t= _for_each2(storage);
		//auto end1 = std::chrono::high_resolution_clock::now();
		//std::chrono::duration<float, std::milli> elapsed1 = end1 - start1;
		//time
		std::cout << "--------------------------------------------------------------"<<std::endl;
		std::cout << "| vector (acces all) "<<std::endl;
		std::cout << "| Time (ms) = "<<  t <<std::endl;//elapsed1.count() <<std::endl;
		std::cout << "--------------------------------------------------------------"<<std::endl;
		t=_for_each_AccessOne2<1,vector_t >(storage);
		std::cout << "--------------------------------------------------------------"<<std::endl;
		std::cout << "| vector (access one)"<<std::endl;
		std::cout << "| Time (ms) = "<< t <<std::endl;//elapsed1.count() <<std::endl;
		std::cout << "--------------------------------------------------------------"<<std::endl;
		for(size_t i=0; i<10; i++)
					std::cout<< storage[i] << std::endl;


	}



}


