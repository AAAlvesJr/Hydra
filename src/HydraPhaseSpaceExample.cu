#include "hip/hip_runtime.h"
/*
 * HydraPhaseSpaceExample.cu
 *
 *  Created on: Sep 22, 2016
 *      Author: augalves
 */

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <time.h>
#include <string>
#include <map>
#include <vector>
#include <array>
#include <tuple>
#include <chrono>
#include <type_traits>
#include <typeinfo>
//command line
#include <tclap/CmdLine.h>
#define CUDA_API_PER_THREAD_DEFAULT_STREAM

//this lib
#include <hydra/Types.h>
#include <hydra/Vector4R.h>
#include <hydra/PhaseSpace.h>
#include <hydra/Containers.h>
#include <hydra/Evaluate.h>
#include <hydra/Function.h>
#include <hydra/FunctorArithmetic.h>
#include <hydra/FunctionWrapper.h>
#include <hydra/Copy.h>


//root
#include <TROOT.h>
#include <TH1D.h>
#include <TF1.h>
#include <TH2D.h>
#include <TH3D.h>
#include <TApplication.h>
#include <TCanvas.h>
#include <TColor.h>
#include <TString.h>
#include <TStyle.h>
#include "RooGlobalFunc.h"
#include "RooRealVar.h"
#include "RooDataSet.h"
#include "RooDataHist.h"
#include "RooCFunction1Binding.h"
#include "RooTFnBinding.h"
#include "RooPlot.h"

#include <src/Gauss.h>
#include <src/Exp.h>

using namespace std;
using namespace hydra;


GInt_t main(int argv, char** argc)
{


	size_t  nentries       = 0;
	GReal_t mother_mass    = 0;
	GReal_t daughter1_mass = 0;
	GReal_t daughter2_mass = 0;
	GReal_t daughter3_mass = 0;


	try {

		TCLAP::CmdLine cmd("Command line arguments for HydraRandomExample", '=');

		TCLAP::ValueArg<GULong_t> NArg("n", "number-of-events",
				"Number of events",
				true, 5e6, "long");
		cmd.add(NArg);

		TCLAP::ValueArg<GReal_t> MassMotherArg("m", "mother-mass",
				"Mass of mother particle",
				true, 0.0, "double");
		cmd.add(MassMotherArg);

		TCLAP::ValueArg<GReal_t> MassDaughter1Arg("a", "daughter-a-mass",
				"Mass of daughter particle 'a' (m -> a b c)",
				true, 0.0, "double");
		cmd.add(MassDaughter1Arg);

		TCLAP::ValueArg<GReal_t> MassDaughter2Arg("b", "daughter-b-mass",
				"Mass of daughter particle 'b' (m -> a b c)",
				true, 0.0, "double");
		cmd.add(MassDaughter2Arg);

		TCLAP::ValueArg<GReal_t> MassDaughter3Arg("c", "daughter-c-mass",
				"Mass of daughter particle 'c' (m -> a b c)",
				true, 0.0, "double");
		cmd.add(MassDaughter3Arg);

		// Parse the argv array.
		cmd.parse(argv, argc);

		// Get the value parsed by each arg.
		nentries       = NArg.getValue();
		mother_mass    = MassMotherArg.getValue();
		daughter1_mass = MassDaughter1Arg.getValue();
		daughter2_mass = MassDaughter2Arg.getValue();
		daughter3_mass = MassDaughter3Arg.getValue();

	}
	catch (TCLAP::ArgException &e)  {
		std::cerr << "error: " << e.error() << " for arg " << e.argId()
														<< std::endl;
	}


	Vector4R B0(mother_mass, 0.0, 0.0, 0.0);
	vector<GReal_t> massesB0{daughter1_mass, daughter2_mass, daughter3_mass };

	/// Create PhaseSpace object for B0-> K pi J/psi
	PhaseSpace<3> phsp(B0.mass(), massesB0);

	Events<3, device> B02JpsiKpi_Events_d(nentries);

	auto start = std::chrono::high_resolution_clock::now();
	phsp.Generate(B0, B02JpsiKpi_Events_d);
	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	//time
	std::cout << "-----------------------------------------"<<std::endl;
	std::cout << "| Time (s) ="<< GReal_t(elapsed.count())/1000000 <<std::endl;
	std::cout << "-----------------------------------------"<<std::endl;

	for( size_t i=0; i<10; i++ ){
		cout << B02JpsiKpi_Events_d[i] << endl;
	}


	auto Weight = [] __host__ __device__ (Events<3, device>::value_type event )
	{ return thrust::get<0>(event) ; };

	auto MB0 = [] __host__ __device__ (Events<3, device>::value_type event )
	{ return (thrust::get<1>(event) + thrust::get<2>(event) +	thrust::get<3>(event) ).mass();	};

	auto M12 = [] __host__ __device__ ( Events<3, device>::value_type event)
	{ return  (thrust::get<1>(event)+ thrust::get<2>(event)).mass2(); };

	auto M13 = [] __host__ __device__( Events<3, device>::value_type event)
	{ return  (thrust::get<1>(event)+ thrust::get<3>(event)).mass2(); };

	auto M23 = [] __host__ __device__( Events<3, device>::value_type event)
	{ return  (thrust::get<2>(event)+ thrust::get<3>(event)).mass2(); };

	auto Weight_W  = LambdaWrapper<GReal_t( Events<3, device>::value_type),
					decltype(Weight) >(Weight);

	auto MB0_W  = LambdaWrapper<GReal_t( Events<3, device>::value_type),
				decltype(MB0) >(MB0);

	auto M12_W  = LambdaWrapper<GReal_t( Events<3, device>::value_type),
			decltype(M12) >(M12);

	auto M13_W  = LambdaWrapper<GReal_t( Events<3, device>::value_type),
				decltype(M13) >(M13);

	auto M23_W  = LambdaWrapper<GReal_t( Events<3, device>::value_type),
					decltype(M23) >(M23);



	auto functors = thrust::make_tuple(Weight_W, MB0_W, M12_W, M13_W, M23_W);
	auto range_0 =make_range( B02JpsiKpi_Events_d.WeightsBegin(), B02JpsiKpi_Events_d.WeightsEnd());
	auto range_1 =make_range( B02JpsiKpi_Events_d.DaughtersBegin(1), B02JpsiKpi_Events_d.DaughtersBegin(1));
	auto range_2 =make_range( B02JpsiKpi_Events_d.DaughtersBegin(2), B02JpsiKpi_Events_d.DaughtersBegin(2));
	auto range_3 =make_range( B02JpsiKpi_Events_d.DaughtersBegin(3), B02JpsiKpi_Events_d.DaughtersBegin(3));

	auto result_d = Eval( functors, range_0,range_1,range_2,range_3 );

	for( size_t i=0; i<10; i++ ){
			cout << result_d[i] << endl;
	}


	auto result_h = get_copy<host>(result_d);

	for( size_t i=0; i<10; i++ ){
				cout << result_h[i] << endl;
		}


	Events<3, host> B02JpsiKpi_Events_h(B02JpsiKpi_Events_d);

	TH2D dalitz("dalitz", ";M(K#pi); M(J/#Psi#pi)", 100,
			pow(daughter2_mass+daughter3_mass,2), pow(mother_mass - daughter1_mass,2),
			100, 	pow(daughter1_mass+daughter3_mass,2), pow(mother_mass - daughter2_mass,2));



	for(auto event: B02JpsiKpi_Events_h){

		GReal_t weight = thrust::get<0>(event);

		Vector4R Jpsi = thrust::get<1>(event);
		Vector4R K    = thrust::get<2>(event);
		Vector4R pi   = thrust::get<3>(event);

		Vector4R Jpsipi = Jpsi + pi;
		Vector4R Kpi    = K + pi;
		GReal_t mass1 = Kpi.mass();
		GReal_t mass2 = Jpsipi.mass();

		dalitz.Fill(mass1*mass1 , mass2*mass2,  weight);
	}





	TApplication *myapp=new TApplication("myapp",0,0);
	TCanvas canvas_gauss("canvas_gauss", "Gaussian distribution", 500, 500);
	dalitz.Draw("colz");
	canvas_gauss.Print("PHSP.pdf");
	myapp->Run();

return 0;
}
